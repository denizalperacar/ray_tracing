#include "common.h"
#include "toojpeg.h"



class color {
public:

	CBRT_HOST_DEVICE color() : r(0), g(0), b(0) {}

	CBRT_HOST_DEVICE color(uint8_t red, uint8_t green, uint8_t blue) 
		: r(red), g(green), b(blue) {}

	CBRT_HOST std::ofstream& draw(std::ofstream& os) {
		os << r << " " << g << " " << b << "\n";
		return os;
	}

	CBRT_HOST_DEVICE void print() {
		printf("%d %d %d\n", r, g, b);
	}

public:
	uint8_t r;
	uint8_t g;
	uint8_t b;

};

std::ostream& operator<<(std::ostream& os, color obj) {
	os << +obj.r << " " << +obj.g << " " << +obj.b << "\n";
	return os;
}

std::fstream& operator<<(std::fstream& os, color obj) {
	os << +obj.r << " " << +obj.g << " " << +obj.b << "\n";
	return os;
}

// solve this 
CBRT_KERNEL void render(color* result) {
	uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t j = threadIdx.y + blockDim.y * blockIdx.y;
	uint32_t idx = j * gridDim.x * blockDim.x + i;

	color c;

	if (i < DEFAULT_IMAGE_WIDTH && j < DEFAULT_IMAGE_HEIGHT) {
		float r = (float)i / (DEFAULT_IMAGE_WIDTH - 1);
		float g = (float)j / (DEFAULT_IMAGE_HEIGHT - 1);
		float b = 0.25;

		c.r = (uint8_t)(255.999 * r);
		c.g = (uint8_t)(255.999 * g);
		c.b = (uint8_t)(255.999 * b);

		result[idx] = c;
		//c.print();
	}
}

class image {

public:
	image(std::string name) {
		file.open(name, std::ios_base::out | std::ios_base::binary);
	}

	~image() {
		file.close();
	}

	void getWriter(unsigned char c) {
		file << c;
	}

private:
	std::ofstream file;
};


// output file
std::ofstream myFile("example.jpg", std::ios_base::out | std::ios_base::binary);

// write a single byte compressed by tooJpeg
void image_output(unsigned char byte)
{
	myFile << byte;
}

int main() {

	color* device_ptr;
	color* host_ptr = (color*)malloc(DEFAULT_IMAGE_WIDTH * DEFAULT_IMAGE_HEIGHT * sizeof(color));


	size_t size = DEFAULT_IMAGE_WIDTH * DEFAULT_IMAGE_HEIGHT * sizeof(color);
	hipMalloc(&device_ptr, size);

	dim3 grid(DEFAULT_IMAGE_WIDTH / NUM_THREADS_MIN, DEFAULT_IMAGE_HEIGHT / NUM_THREADS_MIN);
	dim3 block(NUM_THREADS_MIN, NUM_THREADS_MIN);
	render << < grid, block  >> > (device_ptr);

	hipMemcpy(host_ptr, device_ptr, size, hipMemcpyDeviceToHost);

	void (image:: * pf)(unsigned char c) = &image::getWriter;
	auto ok = TooJpeg::writeJpeg(image_output, host_ptr, DEFAULT_IMAGE_WIDTH, DEFAULT_IMAGE_HEIGHT, true, 90, false);


	hipFree(device_ptr);
	delete [] host_ptr;
	return 0;
}

